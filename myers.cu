#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include "bitVector.cpp"
#include <omp.h>
#include <string.h>
#include <math.h>
#include <bitset>
#include <climits>
#include <vector>
#include <ctype.h>
//#include "../common/common.h"
#include<sys/time.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/device_vector.h> 
#include <thrust/host_vector.h>
using namespace std;
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

class Sequence{
	public:
	    char * name;
	    char * seq;
	    long size;
	    void readSequence(const char * file);
	    
};
	

void Sequence::readSequence(const char * file)
{
	FILE *f1 = fopen(file, "r" );
       	fpos_t position;
	
	char s1[100];    
       	fgets(s1,100,f1);
	name = (char *)malloc(strlen(s1));
        memset(name, 0, strlen(s1));

	for( int i = 0; i < strlen(s1); i++)
	{	
		name[i] = s1[i+1];
	}	

       	fgetpos (f1, &position);
       	int x = ftell(f1);
       	fsetpos (f1, &position);
       	fseek(f1,SEEK_SET , SEEK_END);
       	int y = ftell(f1);
       	size = y-x;
       	seq = new char[size];
       	fsetpos(f1, &position);
       	fread( seq, size, 1, f1 );
	size = strlen(seq);
	for( int i=0 ; i < strlen(seq); ++i )
		seq[i] = toupper( seq[i] ) ;

       	fclose( f1 );
}

__global__ void preprocessing( char * d_characterSet, char * d_vec, char * d_query, int len, int charLength )
{
	int i;
	int j = 0;
	int len1 = len + 1;
        for( i = len-1; i >= 0; i--)
        {
                if( d_characterSet[threadIdx.x] == d_query[i] )
                {
			d_vec[threadIdx.x*len1 + j ] = '1';
                }
                else
                {
                        d_vec[threadIdx.x*len1 + j ] = '0';
                }
		j++;
        }
	d_vec[threadIdx.x*len1+j] = '\0';
}

char*  lookup(char dev_seq, char* char_set, int len)
{
	char * str_ptr;
	str_ptr = (char*)malloc(sizeof(char)*len);
	switch(dev_seq){
	
		case 'A':{
			for (int i = 0;i<len;i++)
				str_ptr[i] = char_set[0*len+i];
			break;
			}
		case 'G':{
			  for (int i = 0;i<len;i++)
                                str_ptr[i] = char_set[1*len+i];
			break;
			}
		case 'C':{
			 for (int i = 0;i<len;i++)
                                str_ptr[i] = char_set[2*len+i];
			break;
			}	
		case 'T':{
			  for (int i = 0;i<len;i++)
                                str_ptr[i] = char_set[3*len+i];
			break;
			}	
		default:
			 memset(str_ptr, '0', sizeof(char)*len);

	}
	return str_ptr;
}

char* binary_add( char opd1[], char opd2[], int len)
{
	int len1 = len + 1;
	char* result;;  // To store the sum bits
	result = (char*)malloc(sizeof(char)*len1); 
 	memset(result, '0', sizeof(char)*len1);
	bool carry = false;  // Initialize carry
	std::string sum = "";
	int charToIntOne;
	int charToIntTwo;
    	for (int i = len-1; i >= 0 ; i--)
    	{
		charToIntOne = opd1[i] - '0';
        	charToIntTwo = opd2[i] - '0';
 
        	if (carry == true && (charToIntOne + charToIntTwo) >= 1)
        	{
            		sum += "0";
        	}
        	else if (carry == true && (charToIntOne + charToIntTwo) == 0)
        	{
            		sum += "1";
            		carry = false;
        	}
        	else
        	{
            		if ((charToIntOne + charToIntTwo) > 1)
            		{
                		sum += "0";
                		carry = true;
            		}
            		else if ((charToIntOne + charToIntTwo) == 1)
            		{
                		sum += "1";
                		carry = false;
            		}
            		else
            		{
                		sum += "0";
                		carry = false;
            		}
        	}
    	}
	int k; 
    	// if overflow, then add a leading 1
    	if (carry == true){
        	sum += "1";
		k = sum.size()-2;
	}
	else{
		k = sum.size()- 1;
	}
	char res[len1];
	//cout<<sum.size()<<"    "<<sum<<endl;
	for (int j = 0; j < len; j++)
    	{
		  res[j] = sum.at(k);
		  k--;
    	}
	res[len +1] = '\0';
	//memset(result,res[len2],sizeof(char)*len2);
	strcpy(result,res);
	//cout<<result<<endl;
    	return result;
}

char* leftShift( char opd[], int len)
{
	char * res;
	res = (char*) malloc((len)*sizeof(char));
	memset(res,'0',len);
	for (int i = 0; i < len; i++) 
	{
		opd[i] = opd[i+1];
		if( i == len - 1)
			opd[i] = '0';	
	}
	strcpy(res,opd);
	return res;
}
__global__ void ORoperation( char * dev_opd1, char * dev_opd2, char * dev_res, int len) 
{
		dev_res[threadIdx.x] = dev_opd1[threadIdx.x] | dev_opd2[threadIdx.x];
	//	printf("%c OR %c = %c\n",dev_opd1[threadIdx.x],dev_opd2[threadIdx.x],dev_res[threadIdx.x]);
}

__global__ void ANDoperation( char* dev_opd1, char* dev_opd2, char* dev_AND, int len)
{
	//	printf("id : %d \n", threadIdx.x);    
		dev_AND[threadIdx.x] = dev_opd1[threadIdx.x] & dev_opd2[threadIdx.x];
	//	printf("%c AND %c = %c\n",dev_opd1[threadIdx.x],dev_opd2[threadIdx.x],dev_AND[threadIdx.x]);
}

__global__ void  XORoperation( char* dev_opd1, char* dev_opd2, char* dev_XOR, int len)
{
		int a = dev_opd1[threadIdx.x] - '0';
		int b = dev_opd2[threadIdx.x] - '0';
		int res = a ^ b;
		dev_XOR[threadIdx.x] = res + '0';
}

__global__ void NOToperation( char* dev_opd, char* dev_NOT, int len)
{
		if( dev_opd[threadIdx.x] == '0' )
			dev_NOT[threadIdx.x] = '1';
		else
			dev_NOT[threadIdx.x] = '0';
}

int main()
{
	//computing Time
	struct timeval t1, t2;

	gettimeofday(&t1, 0);

	int mismatch = 800;
	//Reading the sequences from FASTA files
        Sequence database;
	const char * file = "1000_sequence.txt";
	database.readSequence(file);
	cout<<"Database Details"<<endl;
        cout<<"name : "<<database.name<<endl;
        cout<<"size : "<<database.size<<endl;
       //cout<<"sequence"<<endl<<database.seq;
	
	Sequence query[2];


	const char * file1 = "pattern.txt";
	query[0].readSequence(file1);
        cout<<"Query 0 Details"<<endl;
        cout<<"name : "<<query[0].name<<endl;
        cout<<"size : "<<query[0].size<<endl;
       //cout<<"sequence"<<endl<<query[0].seq;
	
	/*const char * file2 = "query1.txt";
        query[1].readSequence(file2);
        cout<<"Query 1 Details"<<endl;
        cout<<"name : "<<query[1].name<<endl;
        cout<<"size : "<<query[1].size<<endl;
        cout<<"sequence"<<endl<<query[1].seq;*/

	char characterSet[] = "AGCT";
		
	for( int i = 0; i < 1; i++)
	{	//int i = 0;
		char * d_characterSet;
		char * d_query;
		int num_cores = strlen(characterSet);
		int len = query[i].size - 1;
                int len1 = query[i].size;
		char h_vec[num_cores][len1];
		char * d_vec;
		hipMalloc((char**)&d_vec, num_cores*len1*sizeof(char));
		
		hipMalloc((char**)&d_query, len* sizeof(char));
		hipMalloc((char**)&d_characterSet, strlen(characterSet)*sizeof(char));

		hipMemcpy( d_query, query[i].seq, len, hipMemcpyHostToDevice);
		hipMemcpy( d_characterSet, characterSet, strlen(characterSet), hipMemcpyHostToDevice );
		
		dim3 block(num_cores);
		preprocessing <<< 1, num_cores >>> (d_characterSet, d_vec, d_query, len, strlen(characterSet));
		//hipDeviceSynchronize();
		//cudaCheckErrors("error");
		hipMemcpy( h_vec, d_vec, num_cores*len1, hipMemcpyDeviceToHost );
		/*for ( int j = 0;j<num_cores;j++)
		{
			for(int k = 0;k<len1;k++)
			{
				cout<<h_vec[j][k];
			}
			cout<<endl;
		}*/
		
		char VN[len1];
		memset(VN,'0',sizeof(char)*len);
		char VP[len1];
		memset(VP,'1',sizeof(char)*len);
		char X[len];
		
		char D0[len];
                memset(D0,'0',sizeof(char)*len);
		char HN[len];
                memset(HN,'0',sizeof(char)*len);
		char HP[len];
                memset(HP,'0',sizeof(char)*len);
		char temp[len];
		memset(temp,'0',sizeof(char)*len);
		int score = len;
//		begin = clock();
		//cout<<"VP: "<<VP<<endl;
		//cout<<"VN: "<<VN<<endl;
		cout<<"The max score: "<<score<<endl;
		for( int k = 0; k < database.size - 1; k++ )
		{
			char* ptr;

			ptr = lookup( database.seq[k], (char*)h_vec, len1 );
			char Deq[len1];
			strncpy(Deq, ptr, (len1)*sizeof(char));
			//cout<<"******************Reading Text["<<k<<"]*****"<<database.seq[k]<<"****************************************"<<endl;
			//cout<<"Deq: "<<Deq<<endl;
				
			//OR operation on device 
			char* dev_opd1;
			char* dev_opd2;
			char* dev_OR;
			hipMalloc((char**)&dev_opd1,len1*sizeof(char));  
			hipMalloc((char**)&dev_opd2, len1*sizeof(char));
			hipMalloc((char**)&dev_OR, len*sizeof(char));
			hipMemcpy( dev_opd1, Deq, len1, hipMemcpyHostToDevice );
			hipMemcpy( dev_opd2, VN, len1, hipMemcpyHostToDevice );
			
			dim3 block1(len);
			ORoperation <<< 1, len >>> ( dev_opd1, dev_opd2, dev_OR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(X, dev_OR, len, hipMemcpyDeviceToHost );
			/*cout<<"X is  ";
			for( int x=0;x<len;x++)
				cout<<X[x];
			cout<<endl;*/
			
			char* dev_AND;
			char host_AND[len];
			hipMalloc((char**)&dev_AND, len*sizeof(char));
			hipMemcpy( dev_opd2, VP, len, hipMemcpyHostToDevice);
			ANDoperation <<<1,len>>> ( dev_OR, dev_opd2, dev_AND, len);
			hipMemcpy(host_AND, dev_AND, len, hipMemcpyDeviceToHost);
			/*cout<<"AND of X and VP: ";
			for( int x=0;x<len;x++)
                                cout<<host_AND[x];
                        cout<<endl;*/
			
			char* sum_ptr;
			sum_ptr = binary_add( host_AND, VP, len);
			char host_SUM[len1];
                        strncpy(host_SUM, sum_ptr,(len1)*sizeof(char));
			//cout<<"Sum of host_AND & VP : "<<host_SUM<<endl;

			char host_XOR[len];
			char* dev_XOR;
			//memset(host_XOR, '0', sizeof(char) * len);
			hipMalloc((char**)&dev_XOR, len*sizeof(char));
			//hipMalloc((char**)&dev_opd1, len*sizeof(char));
			//hipMemcpy( dev_XOR, host_XOR, len, hipMemcpyHostToDevice);
			hipMemcpy( dev_opd1, host_SUM, len1, hipMemcpyHostToDevice);
			XORoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_XOR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(host_XOR, dev_XOR, len, hipMemcpyDeviceToHost);
			/*cout<<" XOR: ";
			for( int x=0;x<len;x++)
                                cout<<host_XOR[x];
                        cout<<endl;*/

			hipMemcpy( dev_opd2, X, len, hipMemcpyHostToDevice );
			//hipMemcpy( dev_OR, D0, len, hipMemcpyHostToDevice );
			ORoperation <<< 1,len >>> ( dev_XOR, dev_opd2, dev_OR, hipMemcpyHostToDevice);
			hipMemcpy( D0, dev_OR, len, hipMemcpyDeviceToHost);
			/*cout<<" D0: ";
			for( int x=0;x<len;x++)
                                cout<<D0[x];
                        cout<<endl;*/

			hipMalloc((char**)&dev_opd1, len*sizeof(char));
			hipMemcpy( dev_opd1, VP, len, hipMemcpyHostToDevice);
			ANDoperation <<<1,len>>> (dev_opd1, dev_OR, dev_AND, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy( HN, dev_AND, len, hipMemcpyDeviceToHost);
			/*cout<<" HN: ";
			for( int x=0;x<len;x++)
                                cout<<HN[x];
                        cout<<endl;*/
			
			hipMemcpy( dev_opd2, D0, len, hipMemcpyHostToDevice );
			ORoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_OR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(temp, dev_OR, len, hipMemcpyDeviceToHost);
			/*cout<<"OR of D0 and VP: ";  
			for( int x=0;x<len;x++)
                                cout<<temp[x];
                        cout<<endl;*/
		
			char * dev_NOT;
			hipMalloc((char**)&dev_NOT, len*sizeof(char));
			NOToperation <<<1,len>>> ( dev_OR, dev_NOT, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(temp, dev_NOT, len, hipMemcpyDeviceToHost);
			/*cout<<" not: ";
			for( int x=0;x<len;x++)
                                cout<<temp[x];
                        cout<<endl;*/

			hipMemcpy(dev_opd1, VN, len, hipMemcpyHostToDevice);
			ORoperation <<<1,len>>> ( dev_opd1, dev_NOT, dev_OR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(HP, dev_OR, len, hipMemcpyDeviceToHost);
			/*cout<<" HP: ";
			for( int x=0;x<len;x++)
                                cout<<HP[x];
                        cout<<endl;*/

			//score check
			char h_arr[len];
			int ind;
			for( ind = 0; ind < len; ind++ )
			{
				if(ind == 0){
					h_arr[ind] = '1';
				}
				else{
					h_arr[ind] = '0';
				}
			}
			char tmp1[len];
			char tmp2[len];
			/*cout<<"should be 100: ";
			 for( int x=0;x<len;x++)
                                cout<<h_arr[x];
                        cout<<endl;*/	
			hipMemcpy(dev_opd1, h_arr, len, hipMemcpyHostToDevice);
                        hipMemcpy(dev_opd2, HP, len, hipMemcpyHostToDevice);
                        ANDoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_AND, len);
			hipMemcpy(tmp1, dev_AND, len, hipMemcpyDeviceToHost);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
                        
                        /*cout<<"HP AND 100:  ";
			for( int x=0;x<len;x++)
                                cout<<tmp1[x];
                        cout<<endl;*/

                        hipMemcpy(dev_opd2, HN, len, hipMemcpyHostToDevice);
                        ANDoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_AND, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
                        hipMemcpy(tmp2, dev_AND, len, hipMemcpyDeviceToHost);
                        /*cout<<"HN AND 100:  ";
			for( int x=0;x<len;x++)
                                cout<<tmp2[x];
                        cout<<endl;*/
			
			
			bool res1 = false;
			bool res2 = false;
			
			for( int y = 0; y < len; y++ )
			{
				if(tmp1[y] != '0')
				{	
					res1 = true;
					break;
				}
			}			
	
			for( int i = 0; i < len; i++ )
                        {
                                if(tmp2[i] != '0')
                                {
                                        res2 = true;
                                        break;
                                }
                        }
			if( res1 == true)		
				score = score + 1;
			else if( res2 == true)
				score = score - 1;
							
			//cout<<"score: "<<score<<endl;

			if( score <= mismatch){
				//cout<<"appox. match at position: "<<k<<" score: "<<score<<" text character  " <<database.seq[k]<<endl;
			}
			//VN and VP for next column
			char * shft;
                        shft =  leftShift( HP, len);
                        strncpy(X,shft,len*sizeof(char));
                        /*cout<<"new X: ";
                        for( int x=0;x<len;x++)
                                cout<<X[x];
                        cout<<endl;*/

			hipMemcpy(dev_opd1, X, len, hipMemcpyHostToDevice);
			hipMemcpy(dev_opd2, D0, len, hipMemcpyHostToDevice);
			ANDoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_AND, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
			hipMemcpy(VN, dev_AND, len, hipMemcpyDeviceToHost);
			/*cout<<"new VN: ";
			for( int x=0;x<len;x++)
                                cout<<VN[x];
                        cout<<endl;*/

			ORoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_OR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
                        hipMemcpy(temp, dev_OR, len, hipMemcpyDeviceToHost);
                        /*cout<<"OR of X & D0: ";
			for( int x=0;x<len;x++)
                                cout<<temp[x];
                        cout<<endl;*/

			NOToperation <<<1,len>>> ( dev_OR, dev_NOT, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
                        hipMemcpy(temp, dev_NOT, len, hipMemcpyDeviceToHost);
                        /*cout<<" not: ";
			for( int x=0;x<len;x++)
                                cout<<temp[x];
                        cout<<endl;*/

			shft =  leftShift( HN, len);
			char shftHP[len];
                        strncpy(shftHP,shft,len*sizeof(char));
                        //cout<<"shift left of HP: "<<shftHP<<endl;
		
			hipMemcpy(dev_opd1, shftHP, len, hipMemcpyHostToDevice);
                        hipMemcpy(dev_opd2, temp, len, hipMemcpyHostToDevice);
                        ORoperation <<<1,len>>> (dev_opd1, dev_opd2, dev_OR, len);
			//hipDeviceSynchronize();
                	cudaCheckErrors("error");
                        hipMemcpy(VP, dev_OR, len, hipMemcpyDeviceToHost);
                        /*cout<<"new VP: ";
			for( int x=0;x<len;x++)
                                cout<<VP[x];
                        cout<<endl;*/		
		}	
	
	}		
	
	gettimeofday(&t2, 0);

	double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;

	printf("Time to generate:  %3.1f ms \n", time);
//	end = clock();
//	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
 //       printf("Time to generate:  %3.1f \n", time_spent);
	return 0;
}
